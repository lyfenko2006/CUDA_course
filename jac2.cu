#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <time.h>
#include <sys/time.h>

#define a(i, j, k) a[((i) * nn + (j)) * kk + (k)]
#define b(i, j, k) b[((i) * nn + (j)) * kk + (k)]
#define d(i, j, k) d[((i) * nn + (j)) * kk + (k)]

#define Max(a, b) ((a) > (b) ? (a) : (b))

#define L 384
#define ITMAX 100

int ox = 8, oy = 8, oz = 8;
dim3 block = dim3((L + ox - 1) / ox, (L + oy - 1) / oy, (L + oz - 1) / oz);
dim3 thread = dim3(ox, oy, oz);

int i, j, k, it;
double eps;
double MAXEPS = 0.5f;

__global__ void function(int mm, int nn, int kk, double* a, double* b)
{
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int i = blockIdx.z * blockDim.z + threadIdx.z;
	if (i > 0 && i < mm - 1)
		if (j > 0 && j < nn - 1)
			if (k > 0 && k < kk - 1)
				b(i, j, k) = (a(i - 1, j, k) + a(i + 1, j, k) + a(i, j - 1, k) + a(i, j + 1, k)
					+ a(i, j, k - 1) + a(i, j, k + 1)) / 6.;
}

__global__ void difference(int mm, int nn, int kk, double* a, double* b, double *d)
{
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int i = blockIdx.z * blockDim.z + threadIdx.z;
	if (i > 0 && i < mm - 1)
		if (j > 0 && j < nn - 1)
			if (k > 0 && k < kk - 1)
				d(i, j, k) = fabs(a(i, j, k) - b(i, j, k));
}

__global__ void ab(int mm, int nn, int kk, double* a, double* b)
{
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int i = blockIdx.z * blockDim.z + threadIdx.z;
	if (i > 0 && i < mm - 1)
		if (j > 0 && j < nn - 1)
			if (k > 0 && k < kk - 1)
				a(i, j, k) = b(i, j, k);
}

double jac_parallel(double* a, int mm, int nn, int kk, int itmax, double maxeps)
{
	double* b;

	hipMalloc((void**)&b, mm * nn * kk * sizeof(double));
	
	for (it = 1; it <= itmax - 1; it++)
	{
		function << <block, thread >> > (mm, nn, kk, a, b);	
	    hipDeviceSynchronize();
		eps = 0.;

		thrust::device_vector<double> diff(mm * nn * kk);
		double* ptrdiff = thrust::raw_pointer_cast(&diff[0]);
		difference << <block, thread >> > (mm, nn, kk, a, b, ptrdiff);
        hipDeviceSynchronize();

		eps = thrust::reduce(diff.begin(), diff.end(), 0.0, thrust::maximum<double>());
		ab << <block, thread >> > (mm, nn, kk, a, b);
        hipDeviceSynchronize();

		//if (TRACE && it % TRACE == 0)
			printf(" IT = %4i   EPS = %14.7E\n", it, eps);

		if (eps < maxeps)
			break;
	}
	hipFree(b);

	return eps;
}

__global__ void initial(int mm, int nn, int kk, double* a)
{
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int i = blockIdx.z * blockDim.z + threadIdx.z;
	//printf("%lf\n", a(i, j, k));
	if (i >= 0 && i < mm)
		if (j >= 0 && j < nn)
			if (k >= 0 && k < kk)
				if (i == 0 || j == 0 || k == 0 || i == mm - 1 || j == nn - 1 || k == kk - 1)
                   	 		a(i, j, k) = 0;
                		else
                    		a(i, j, k) = 4 + i + j + k;
}

void initial_seq(int mm, int nn, int kk, double *a)
{
    for (i = 0; i < mm; i++)
        for (j = 0; j < nn; j++)
            for (k = 0; k < kk; k++)
            {
                if (i == 0 || j == 0 || k == 0 || i == mm - 1 || j == nn - 1 || k == kk - 1)
                    a(i, j, k) = 0;
                else
                    a(i, j, k) = 4 + i + j + k;
            }
}

double jac_sequence(double* a, int mm, int nn, int kk, int itmax, double maxeps)
{
    double *b = (double*) malloc(nn * mm * kk * sizeof(double));
    double eps;

    /* iteration loop */
    for (it = 1; it <= ITMAX - 1; it++)
    {
        for (i = 1; i < mm - 1; i++)
            for (j = 1; j < nn - 1; j++)
                for (k = 1; k < kk - 1; k++)
                    b(i, j, k) = (a(i - 1, j, k) + a(i, j - 1, k) + a(i, j, k - 1) + a(i, j, k + 1) + a(i, j + 1, k) + a(i + 1, j, k)) / 6.0f;
        
        eps = 0.;
        
        for (i = 1; i < mm - 1; i++)
            for (j = 1; j < nn - 1; j++)
                for (k = 1; k < kk - 1; k++)
                {
                    double tmp = fabs(b(i, j, k) - a(i, j, k));
                    eps = Max(tmp, eps);
                    a(i, j, k) = b(i, j, k);
                }

        printf(" IT = %4i   EPS = %14.7E\n", it, eps);
        if (eps < MAXEPS)
            break;
    }
    free(b);
    return eps;
}

void print_benchmark(double eps, struct timeval startt, struct timeval endt)
{
    printf(" Jacobi3D Benchmark Completed.\n");
    printf(" Size            = %4d x %4d x %4d\n", L, L, L);
    printf(" Iterations      =       %12d\n", ITMAX);
    printf(" Time in seconds =       %12.2lf\n", endt.tv_sec - startt.tv_sec + (endt.tv_usec - startt.tv_usec) * 0.000001);
    printf(" Operation type  =     floating point\n");
    printf(" Verification    =       %12s\n", (fabs(eps - 5.058044) < 1e-11 ? "SUCCESSFUL" : "UNSUCCESSFUL"));
    //printf(" Verification    =       %12s\n", (eps <= 5.058044  ? "SUCCESSFUL" : "UNSUCCESSFUL"));

    printf(" END OF Jacobi3D Benchmark\n");
}

int main(void)
{
	double *a;
    double eps;

    struct timeval startt, endt;

	hipMalloc((void**)&a, L * L * L * sizeof(double));
	initial << <block, thread >> > (L, L, L, a);

    gettimeofday(&startt, NULL);
	eps = jac_parallel(a, L, L, L, ITMAX, MAXEPS);
    hipDeviceSynchronize();
    gettimeofday(&endt, NULL);

	hipFree(a);
    
    print_benchmark(eps, startt, endt);

    a = (double*) malloc(L * L * L * sizeof(double));
    initial_seq(L, L, L, a);

    gettimeofday(&startt, NULL);
    eps = jac_sequence(a, L, L, L, ITMAX, MAXEPS);
    gettimeofday(&endt, NULL);

    print_benchmark(eps, startt, endt);

    free(a);
	return 0;
}
